#include "cudaImmediary.h"
#include "mandelbrot-kernel.cuh"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <iomanip>
#define CudaErrorCheck(var, function) \
var = function; \
if(_CudaErrorCheck(error)) { return 1; }

bool _CudaErrorCheck(hipError_t const& error)
{
  if (error != hipSuccess)
  {
    fprintf(stderr, "%s>%s\n", hipGetErrorName(error), hipGetErrorString(error));
    return 1;
  }
  return 0;
}
int DivUp(int a, int b) {
  return ((a % b) != 0) ? (a / b + 1) : (a / b);
}
int InitializeCudaData(cuMandelbrotData* cuData, const size_t buffer_size)
{
  hipError_t error;
  CudaErrorCheck(error, hipMalloc(&(cuData->cuBuffer), buffer_size));
  CudaErrorCheck(error, hipMalloc(&(cuData->cuMandel), sizeof(MandelbrotPlot_t)));
  cuData->init = true;
  cuData->buffer_size = buffer_size;
  return 0;
}
int UpdateCudaData(cuMandelbrotData& cuData, MandelbrotPlot_t const& plot)
{
  hipError_t error;
  const unsigned PLOT_SIZE = (plot.height * plot.width * 4);
  if ((cuData.buffer_size) != PLOT_SIZE)
  {
    CudaErrorCheck(error, hipFree(cuData.cuBuffer));
    CudaErrorCheck(error, hipMalloc(&(cuData.cuBuffer), PLOT_SIZE));
    cuData.buffer_size = PLOT_SIZE;
  }
  CudaErrorCheck(error, hipMemcpy(cuData.cuMandel, &plot, sizeof(MandelbrotPlot_t), hipMemcpyHostToDevice));
  return 0;
}
uint8_t* MandelbrotCudaCall(
  uint8_t * buffer,
  MandelbrotPlot_t const & plot,
  cuMandelbrotData & cuBuffers
)
{
  buffer[0] =0xD;
  if (cuBuffers.init != true)
    InitializeCudaData(&cuBuffers, plot.height * plot.width * 4);
  UpdateCudaData(cuBuffers, plot);
  dim3 bs(64, 4), grid(DivUp(plot.width, bs.x), DivUp(plot.height, bs.y));
  MandelbrotKernel <<<grid, bs >>> (cuBuffers.cuBuffer, cuBuffers.cuMandel);
  hipMemcpy(buffer, cuBuffers.cuBuffer, cuBuffers.buffer_size, hipMemcpyDeviceToHost);
  return NULL;
}
